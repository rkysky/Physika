#include "hip/hip_runtime.h"
//atomicAdd(XX, 1) -> atomicInc !!!!!!!
#define OUTPUT_TXT

// CUDA Runtime
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <assert.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA SDK samples
#include <hip/hip_runtime_api.h>       // helper for CUDA error checking

#include "Physika_Collision/data_struct/vec3.cuh"
#include "Physika_Collision/data_struct/tools.cuh"
#include "Physika_Collision/data_struct/box.cuh"
#include "Physika_Collision/data_struct/tri3f.cuh"
#include "Physika_Collision/bvh/bvh.cuh"
#include "Physika_Collision/data_struct/pair.cuh"
#include "Physika_Collision/tri-contact/tri-contact.cuh"

#include <math.h>
#include <stdarg.h>

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>

#include <string>

using namespace std;

typedef unsigned int uint;


typedef struct {
	uint  numFace, numVert;
	REAL3 *_dx, *_dx0;
	tri3f *_df;
	g_box *_dfBx;

	// init function
	void init()
	{
		numFace = 0;
		numVert = 0;
		_dx0 = _dx = NULL;
		_df = NULL;
		_dfBx = NULL;
	}

	void destroy()
	{
		if (_dx == NULL) return;

		checkCudaErrors(hipFree(_dx));
		checkCudaErrors(hipFree(_dx0));
		checkCudaErrors(hipFree(_df));
		checkCudaErrors(hipFree(_dfBx));
	}

	void computeWSdata(REAL thickness, bool ccd);
} g_mesh;

//=======================================================

hipDeviceProp_t deviceProp;
extern void initPairsGPU();

void initGPU()
{
	int devID = 0;
	hipGetDevice(&devID);
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

	initPairsGPU();
}

//=======================================================

g_mesh theCloth;
g_bvh* theBVH;
g_front theFront;
g_pair thePairs[2]; // potentially colliding pairs
g_pair retPairs; //results

//rky
int BVH_NUM;
//=======================================================
//rky
void init(int num){
	BVH_NUM = num;
	theBVH=new g_bvh[num];
	//hipMalloc((void **)&theFront, num*sizeof(g_front));
	theFront.init();
}
//rky
void front_clear() {
	theFront.clear();
}

void initPairsGPU()
{
	//pairs[0].init(MAX_PAIR_NUM); // MAX_PAIR_NUM);
	thePairs[1].init(MAX_PAIR_NUM);
	retPairs.init(MAX_PAIR_NUM / 10);
}

void pushMesh2GPU(int  numFace, int numVert, void *faces, void *nodes)
{
	theCloth.init();

	theCloth.numFace = numFace;
	theCloth.numVert = numVert;

	hipMalloc((void **)&theCloth._df, numFace*sizeof(tri3f));
	hipMalloc((void **)&theCloth._dfBx, numFace*sizeof(g_box));
	hipMalloc((void **)&theCloth._dx, numVert*sizeof(REAL3));
	hipMalloc((void **)&theCloth._dx0, numVert*sizeof(REAL3));

	hipMemcpy(theCloth._df, faces, sizeof(tri3f)*numFace, hipMemcpyHostToDevice);
	hipMemcpy(theCloth._dx, nodes, sizeof(REAL3)*numVert, hipMemcpyHostToDevice);
	hipMemcpy(theCloth._dx0, theCloth._dx, sizeof(REAL3)*numVert, hipMemcpyDeviceToDevice);

	theCloth.computeWSdata(0, false);
}

void updateMesh2GPU(void *nodes)
{
	hipMemcpy(theCloth._dx0, theCloth._dx, sizeof(REAL3)*theCloth.numVert, hipMemcpyDeviceToDevice);
	hipMemcpy(theCloth._dx, nodes, sizeof(REAL3)*theCloth.numVert, hipMemcpyHostToDevice);
	theCloth.computeWSdata(0, false);
}

//=======================================================

void pushBVHIdx(int max_level, unsigned int *level_idx, int i)
{
	theBVH[i]._max_level = max_level;
	theBVH[i]._level_idx = new uint[max_level];
	memcpy(theBVH[i]._level_idx, level_idx, sizeof(uint)*max_level);
}

void pushBVH(unsigned int length, int *ids, int i)
{
	theBVH[i]._num = length;
	checkCudaErrors(hipMalloc((void**)&theBVH[i]._bvh, length*sizeof(int) * 2));
	checkCudaErrors(hipMemcpy(theBVH[i]._bvh, ids, length*sizeof(int) * 2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&theBVH[i]._bxs, length*sizeof(g_box)));
	checkCudaErrors(hipMemset(theBVH[i]._bxs, 0, length*sizeof(g_box)));
	theBVH[i].hBxs = NULL;

	//rky
	theBVH[i]._triBxs = theCloth._dfBx;
	//theBVH[i]._triBxs =NULL;
	theBVH[i]._triCones = NULL;
}

void pushBVHLeaf(unsigned int length, int *idf, int i)
{
	checkCudaErrors(hipMalloc((void**)&theBVH[i]._bvh_leaf, length*sizeof(int)));
	checkCudaErrors(hipMemcpy(theBVH[i]._bvh_leaf, idf, length*sizeof(int), hipMemcpyHostToDevice));
}

//======================================================


void refitBVH_Serial(int bvh_id, int length)
{

	refit_serial_kernel << <1, 1, 0 >> >
		(theBVH[bvh_id]._bvh, theBVH[bvh_id]._bxs, theBVH[bvh_id]._triBxs,
		theBVH[bvh_id]._cones, theBVH[bvh_id]._triCones,
		length == 0 ? theBVH[bvh_id]._num : length);

	getLastCudaError("refit_serial_kernel");
	hipDeviceSynchronize();
}

void refitBVH_Parallel(int bvh_id, int st, int length)
{
	BLK_PAR(length);

	refit_kernel << < B, T >> >
		(theBVH[bvh_id]._bvh, theBVH[bvh_id]._bxs, theBVH[bvh_id]._triBxs,
		theBVH[bvh_id]._cones, theBVH[bvh_id]._triCones,
		st, length);

	getLastCudaError("refit_kernel");
	hipDeviceSynchronize();
}

//rky
void refitBVH(int bvh_id)
{
	// before refit, need to get _tri_boxes !!!!
	// copying !!!
	for (int i = theBVH[bvh_id]._max_level - 1; i >= 0; i--) {
		int st = theBVH[bvh_id]._level_idx[i];
		int ed = (i != theBVH[bvh_id]._max_level - 1) ?
			theBVH[bvh_id]._level_idx[i + 1] - 1 : theBVH[bvh_id]._num - 1;

		int length = ed - st + 1;
		if (i < 5) {
			refitBVH_Serial(bvh_id, length + st);
			break;
		}
		else
		{
			refitBVH_Parallel(bvh_id, st, length);
		}
	}
}


void refitBVH()
{
	// before refit, need to get _tri_boxes !!!!
	// copying !!!
	for (int k = 0; k < BVH_NUM; k++){
		int bvh_id = k;
		for (int i = theBVH[bvh_id]._max_level - 1; i >= 0; i--) {
			int st = theBVH[bvh_id]._level_idx[i];
			int ed = (i != theBVH[bvh_id]._max_level - 1) ?
				theBVH[bvh_id]._level_idx[i + 1] - 1 : theBVH[bvh_id]._num - 1;

			int length = ed - st + 1;
			if (i < 5) {
				refitBVH_Serial(bvh_id, length + st);
				break;
			}
			else
			{
				refitBVH_Parallel(bvh_id, st, length);
			}
		}
	}
}

//===============================================

void pushFront(int num, unsigned int *data,unsigned int *data_id)
{
	g_front *f = &theFront;

	//rky
	//f->init();
	f->push(num, (uint4 *)data,data_id);
}

//===============================================
// show memory usage of GPU
void  reportMemory(char *tag)
{
	//return;

#ifdef OUTPUT_TXT
	size_t free_byte;
	size_t total_byte;
	hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

	if (hipSuccess != cuda_status) {
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
		exit(1);
	}

	REAL free_db = (REAL)free_byte;
	REAL total_db = (REAL)total_byte;
	REAL used_db = total_db - free_db;
	printf("%s: GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
		tag, used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
#endif
}

//===============================================

#define STACK_SIZE 50
#define EMPTY (nIdx == 0)

#define PUSH_PAIR(nd1, nd2)  {\
	nStack[nIdx].x = nd1;\
	nStack[nIdx].y = nd2;\
	nIdx++;\
}

#define POP_PAIR(nd1, nd2) {\
	nIdx--;\
	nd1 = nStack[nIdx].x;\
	nd2 = nStack[nIdx].y;\
}

#define NEXT(n1, n2) 	POP_PAIR(n1, n2)


inline __device__ void pushToFront(int a, int b, uint4 *front, uint *idx, uint ptr)
{
	//	(*idx)++;
	if (*idx < MAX_FRONT_NUM)
	{
		uint offset = atomicAdd(idx, 1);
		front[offset] = make_uint4(a, b, 0, ptr);
	}
}

inline __device__ void sproutingAdaptive(int left, int right,
	int *bvhA, g_box *bxsA, int *bvhB, g_box *bxsB,
	uint4 *front, uint *frontIdx,
	uint2 *pairs, uint *pairIdx, bool update, uint ptr)
{
	uint2 nStack[STACK_SIZE];
	uint nIdx = 0;

	for (int i = 0; i<4; i++)
	{
		if (isLeaf(left, bvhA) && isLeaf(right, bvhB)) {
			pushToFront(left, right, front, frontIdx, ptr);
		}
		else {
			if (!overlaps(left, right, bxsA, bxsB)) {
				pushToFront(left, right, front, frontIdx, ptr);
			}
			else {
				if (isLeaf(left, bvhA)) {
					PUSH_PAIR(left, getLeftChild(right, bvhB));
					PUSH_PAIR(left, getRightChild(right, bvhB));
				}
				else {
					PUSH_PAIR(getLeftChild(left, bvhA), right);
					PUSH_PAIR(getRightChild(left, bvhA), right);
				}
			}
		}

		if (EMPTY)
			return;

		NEXT(left, right);
	}

	while (!EMPTY) {
		NEXT(left, right);
		pushToFront(left, right, front, frontIdx, ptr);
	}
}

inline __device__ void sprouting(int left, int right,
	int *bvhA, g_box *bxsA, int *bvhB, g_box *bxsB,
	uint4 *front, uint *frontIdx,
	int2 *pairs, uint *pairIdx, bool update, uint ptr)
{
	uint2 nStack[STACK_SIZE];
	uint nIdx = 0;

	while (1)
	{
		if (isLeaf(left, bvhA) && isLeaf(right, bvhB)) {
			if (update)
				pushToFront(left, right, front, frontIdx, ptr);

			if (overlaps(left, right, bxsA, bxsB))
				addPair(getTriID(left, bvhA), getTriID(right, bvhB), pairs, pairIdx);
		}
		else {
			if (!overlaps(left, right, bxsA, bxsB)) {
				if (update)
					pushToFront(left, right, front, frontIdx, ptr);

			}
			else {
				if (isLeaf(left, bvhA)) {
					PUSH_PAIR(left, getLeftChild(right, bvhB));
					PUSH_PAIR(left, getRightChild(right, bvhB));
				}
				else {
					PUSH_PAIR(getLeftChild(left, bvhA), right);
					PUSH_PAIR(getRightChild(left, bvhA), right);
				}
			}
		}

		if (EMPTY)
			return;

		NEXT(left, right);
	}
}

__device__ void doPropogate(
	uint4 *front,uint *bvh_id,g_bvh *bvh, uint *frontIdx, int num,
	int2 *pairs, uint *pairIdx, bool update, tri3f *Atris, int idx, bool *flags)
{
	uint4 node = front[idx];

	uint _bvhid[2];
	_bvhid[0] = bvh_id[idx*2];
	_bvhid[1] = bvh_id[idx*2 + 1];

	if (node.z != 0) {
#if defined(_DEBUG) || defined(OUTPUT_TXT)
		atomicAdd(frontIdx + 1, 1);
#endif
		return;
	}

#ifdef USE_NC
	if (flags != NULL && flags[node.w] == 0) {
#if defined(_DEBUG) || defined(OUTPUT_TXT)
		atomicAdd(frontIdx + 2, 1);
#endif
		return;
	}
#endif


	uint left = node.x;
	uint right = node.y;
	int *bvhA = bvh[_bvhid[0]]._bvh;
	g_box *bxsA = bvh[_bvhid[0]]._bxs;
	int *bvhB = bvh[_bvhid[1]]._bvh;
	g_box *bxsB = bvh[_bvhid[1]]._bxs;

	if (isLeaf(left, bvhA) && isLeaf(right, bvhB)) {
		if (overlaps(left, right, bxsA, bxsB))
			if (_bvhid[0] != _bvhid[1])
				addPair(getTriID(left, bvhA), getTriID(right, bvhB), pairs, pairIdx);
			else { // for self ccd, we need to remove adjacent triangles, they will be processed seperatedly with orphan set
				if (!covertex(getTriID(left, bvhA), getTriID(right, bvhB), Atris))
					addPair(getTriID(left, bvhA), getTriID(right, bvhB), pairs, pairIdx);
			}
			return;
	}

	if (!overlaps(left, right, bxsA, bxsB))
		return;

	if (update)
		front[idx].z = 1;

	int ptr = node.w;
	if (isLeaf(left, bvhA)) {
		sprouting(left, getLeftChild(right, bvhB), bvhA, bxsA, bvhB, bxsB, front, frontIdx, pairs, pairIdx, update, ptr);
		sprouting(left, getRightChild(right, bvhB), bvhA, bxsA, bvhB, bxsB, front, frontIdx, pairs, pairIdx, update, ptr);
	}
	else {
		sprouting(getLeftChild(left, bvhA), right, bvhA, bxsA, bvhB, bxsB, front, frontIdx, pairs, pairIdx, update, ptr);
		sprouting(getRightChild(left, bvhA), right, bvhA, bxsA, bvhB, bxsB, front, frontIdx, pairs, pairIdx, update, ptr);
	}
}


__global__ void kernelPropogate(uint4 *front, uint *bvh_id,g_bvh *bvh, uint *frontIdx, int num,
	int2 *pairs, uint *pairIdx, bool update, tri3f *Atris, int stride, bool *flags)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	for (int i = 0; i<stride; i++) {
		int j = idx*stride + i;
		if (j >= num)
			return;

		doPropogate(front,bvh_id, bvh,frontIdx, num,
		 pairs, pairIdx, update, Atris, j, flags);
	}
}

int g_front::propogate(bool &update, bool ccd)
{
	uint dummy[1];
	cutilSafeCall(hipMemcpy(dummy, _dIdx, 1 * sizeof(uint), hipMemcpyDeviceToHost));
#ifdef OUTPUT_TXT
	//printf("Before propogate, length = %d\n", dummy[0]);
#endif

#if defined(_DEBUG) || defined(OUTPUT_TXT)
	uint dummy2[5] = { 0, 0, 0, 0, 0 };
	cutilSafeCall(hipMemcpy(_dIdx + 1, dummy2, 5 * sizeof(int), hipMemcpyHostToDevice));
#endif

	if (dummy[0] != 0) {
		//rky
		//g_bvh *pb1 = &theBVH[id1];
		//g_bvh *pb2 = &theBVH[id2];
		tri3f *faces = theCloth._df;

		int stride = 4;
#ifdef FIX_BT_NUM
		BLK_PAR2(dummy[0], stride);
#else
		BLK_PAR3(dummy[0], stride, getBlkSize((void *)kernelPropogate));
#endif
		g_bvh* _thebvh;
		cutilSafeCall(hipMalloc((void**)&_thebvh, BVH_NUM*sizeof(g_bvh)));
		cutilSafeCall(hipMemcpy(_thebvh, theBVH, BVH_NUM*sizeof(g_bvh),hipMemcpyHostToDevice));
			
		//rky
		kernelPropogate << < B, T >> >
			(_dFront,bvh_id,_thebvh, _dIdx, dummy[0],
			thePairs[1]._dPairs, thePairs[1]._dIdx, update, faces, stride,  theBVH[0]._ctFlags );
		//thePairs[self]._dPairs, thePairs[self]._dIdx, update, faces, stride, (self && !ccd) ? theBVH[1]._ctFlags : NULL);

		hipDeviceSynchronize();
		getLastCudaError("kernelPropogate");
	}

	cutilSafeCall(hipMemcpy(dummy, _dIdx, 1 * sizeof(uint), hipMemcpyDeviceToHost));
#ifdef OUTPUT_TXT
	//printf("After propogate, length = %d\n", dummy[0]);
#endif

#if defined(_DEBUG) || defined(OUTPUT_TXT)
	cutilSafeCall(hipMemcpy(dummy2, _dIdx + 1, 5 * sizeof(int), hipMemcpyDeviceToHost));
	//printf("Invalid = %d, NC culled = %d\n", dummy2[0], dummy2[1]);
#endif

	if (update && dummy[0] > SAFE_FRONT_NUM) {
		printf("Too long front, stop updating ...\n");
		update = false;
	}

	if (dummy[0] > MAX_FRONT_NUM) {
		printf("Too long front, exiting ...\n");
		exit(0);
	}
	return dummy[0];
}

//===============================================

__global__ void
kernel_face_ws(tri3f *face, REAL3 *x, REAL3 *ox, g_box *bxs, bool ccd, REAL thickness, int num)
{
	LEN_CHK(num);

	int id0 = face[idx].id0();
	int id1 = face[idx].id1();
	int id2 = face[idx].id2();

	REAL3 ox0 = ox[id0];
	REAL3 ox1 = ox[id1];
	REAL3 ox2 = ox[id2];
	REAL3 x0 = x[id0];
	REAL3 x1 = x[id1];
	REAL3 x2 = x[id2];

	bxs[idx].set(ox0, ox1);
	bxs[idx].add(ox2);

	if (ccd) {
		bxs[idx].add(x0);
		bxs[idx].add(x1);
		bxs[idx].add(x2);
	}
	//else
	bxs[idx].enlarge(thickness);
}

void g_mesh::computeWSdata(REAL thickness, bool ccd)
{
	if (numFace == 0)
		return;

	{
		int num = numFace;
		BLK_PAR(num);
		kernel_face_ws << <B, T >> > (
			_df, _dx, _dx, _dfBx, ccd, thickness, num);
		getLastCudaError("kernel_face_ws");
	}
}

//===============================================

__global__ void kernelGetCollisions(
	int2 *pairs, int num, 
	REAL3 *cx, tri3f *ctris, int2 *pairRets, uint *pairIdx,
	int stride)
{
	int idxx = blockDim.x * blockIdx.x + threadIdx.x;

	for (int i = 0; i<stride; i++) {

		int j = idxx*stride + i;
		if (j >= num)
			return;

		int idx = j;

		int2 pair = pairs[idx];
		int fid1 = pair.x;
		int fid2 = pair.y;

		tri3f t1 = ctris[fid1];
		tri3f t2 = ctris[fid2];

#ifdef FOR_DEBUG
		bool find = false;
		if (fid1 == 369 && fid2 == 3564)
			find = true;
		if (fid2 == 369 && fid1 == 3564)
			find = true;
#endif

		REAL3 p0 = cx[t1.id0()];
		REAL3 p1 = cx[t1.id1()];
		REAL3 p2 = cx[t1.id2()];
		REAL3 q0 = cx[t2.id0()];
		REAL3 q1 = cx[t2.id1()];
		REAL3 q2 = cx[t2.id2()];

#ifdef FOR_DEBUG
		if (find) {
			printf("%d: %lf, %lf, %lf\n", t1.id0(), p0.x, p0.y, p0.z);
			printf("%d: %lf, %lf, %lf\n", t1.id1(), p1.x, p1.y, p1.z);
			printf("%d: %lf, %lf, %lf\n", t1.id2(), p2.x, p2.y, p2.z);
			printf("%d: %lf, %lf, %lf\n", t2.id0(), q0.x, q0.y, q0.z);
			printf("%d: %lf, %lf, %lf\n", t2.id1(), q1.x, q1.y, q1.z);
			printf("%d: %lf, %lf, %lf\n", t2.id2(), q2.x, q2.y, q2.z);
		}
#endif
		

		if (tricontact::tri_contact(p0, p1, p2, q0, q1, q2))
			//if (fid1 > fid2)
				//addPair(fid2, fid1, pairRets, pairIdx);
			//else
				addPair(fid1, fid2, pairRets, pairIdx);
	}
}

//===============================================

int g_pair::getCollisions(bool self, g_pair &ret)
{
	int num = length();

#ifdef OUTPUT_TXT
	if (self)
		//printf("self pair = %d\n", num);
	//else
		//printf("inter-obj pair = %d\n", num);
#endif

	if (num == 0)
		return 0;

	ret.clear();
	
	int stride = 4;
#ifdef FIX_BT_NUM
	BLK_PAR3(num, stride, 32);
#else
	BLK_PAR3(num, stride, getBlkSize((void *)kernelGetCollisions));
#endif


	kernelGetCollisions << < B, T >> >(_dPairs, num,
		theCloth._dx, theCloth._df, ret._dPairs, ret._dIdx,stride);

	getLastCudaError("kernelGetCollisions");

	int len = ret.length();
#ifdef OUTPUT_TXT
	//printf("collision num = %d\n", len);
#endif

	return len;
}
//===============================================

int getCollisionsGPU(int *rets)
{
	bool update = false;
	int len = 0;

	TIMING_BEGIN
	thePairs[1].clear();

	refitBVH();

	theFront.propogate(update,false);
	hipDeviceSynchronize();
	

	len = thePairs[1].getCollisions(true, retPairs);
	hipDeviceSynchronize();

	TIMING_END("$$$get_collisions_gpu")

	if (len > 0) {
		cutilSafeCall(hipMemcpy(rets, retPairs._dPairs, sizeof(uint)*2*len, hipMemcpyDeviceToHost));
	}

	return len;
}